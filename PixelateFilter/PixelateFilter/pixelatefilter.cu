#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <omp.h>

#define FAILURE 0
#define SUCCESS !FAILURE

#define USER_NAME "acp18dj"		//my user name

void print_help();
int process_command_line(int argc, char *argv[], int *c, char *input_file, char *output_file, char *ppm_format);
int is_two_n(int num);
int image_input(char *input_file, int c, int *width, int *height, char *header, char *output_file, char *ppm_format);
int read_header(FILE *fp, int c, int *width, int *height, char *header, char *output_file, char *ppm_format, char *format);
void cul_average_cpu(int c, int width, int height, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b);
void cul_average_openmp(int c, int width, int height, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b);
void launch_cuda_1D(int c, int width, int height, unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b);
//void launch_cuda_2D(int c, int width, int height, unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b);
void transform_2D_to_1D(unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, int width, int height);
void transform_1D_to_2D(unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, int width, int height);
int image_output(int width, int height, char *ppm_format, char *header, char *output_file);
void checkCUDAError(const char *msg);

typedef enum MODE { CPU, OPENMP, CUDA, ALL } MODE;

MODE execution_mode = CPU;
unsigned char **image_r, **image_g, **image_b;

//texture<unsigned char, hipTextureType2D> texData_r;
//texture<unsigned char, hipTextureType2D> texData_g;
//texture<unsigned char, hipTextureType2D> texData_b;

__device__ unsigned long long int average_r, average_g, average_b;


__global__ void avgKernel_1D(uchar3 *image, const int width, const int height, const int c) {
	// declare share memory for calculating average across the threads
	
	__shared__ unsigned long long int sdata_r, sdata_g, sdata_b;
	//__shared__ float3 sdata;

	// the local variables for summerising values for certain thread
	uchar3 pixel;
	unsigned long long int sum_r = 0, sum_g = 0, sum_b = 0;

	// how many row should be pass for one thread
	int colPerThread = (c > 1024) ? c / 1024 : 1;

	// the imcompleted block index and its width and height 
	int block_x = -1, block_y = -1, rest_x = 0, rest_y = 0;
	if (width % c != 0) {
		block_x = width / c;
		rest_x = width % c;
	}
	if (height % c != 0) {
		block_y = height / c;
		rest_y = height % c;
	}

	// over the width boundary
	if (blockIdx.x != block_x || threadIdx.x < rest_x) {
		for (unsigned int j = 0; j < colPerThread; j++) {
			for (unsigned int i = 0; i < c; i++) {
				// over the height boundary
				if (blockIdx.y == block_y && i >= rest_y) continue;
				if ((blockIdx.x * c + threadIdx.x + j * 1024) >= width) continue;

				unsigned int offset = (blockIdx.y * width * c) + (blockIdx.x * c + threadIdx.x + j * 1024) + (width * i);
				pixel = image[offset];
				/*atomicAdd(&sdata_r, gpu_r[offset]);
				atomicAdd(&sdata_g, gpu_g[offset]);
				atomicAdd(&sdata_b, gpu_b[offset]);*/

				// summerise a the thread
				sum_r += pixel.x;
				sum_g += pixel.y;
				sum_b += pixel.z;
			}
		}
		// summerise within a block 
		/*atomicAdd(&sdata.x, sum_r);
		atomicAdd(&sdata.y, sum_g);
		atomicAdd(&sdata.z, sum_b);*/
		atomicAdd(&sdata_r, sum_r);
		atomicAdd(&sdata_g, sum_g);
		atomicAdd(&sdata_b, sum_b);
	}

	__syncthreads();
	if (threadIdx.x == 0) {
		// summerise the values of all blocks
		/*atomicAdd(&average_r, sdata.x);
		atomicAdd(&average_g, sdata.y);
		atomicAdd(&average_b, sdata.z);*/
		atomicAdd(&average_r, sdata_r);
		atomicAdd(&average_g, sdata_g);
		atomicAdd(&average_b, sdata_b);

		// calculate the average with different size
		if (blockIdx.x == block_x && blockIdx.y == block_y) {
			/*sdata.x /= rest_x * rest_y;
			sdata.y /= rest_x * rest_y;
			sdata.z /= rest_x * rest_y;*/
			sdata_r /= rest_x * rest_y;
			sdata_g /= rest_x * rest_y;
			sdata_b /= rest_x * rest_y;
		}
		else if (blockIdx.x == block_x && blockIdx.y != block_y) {
			/*sdata.x /= rest_x * c;
			sdata.y /= rest_x * c;
			sdata.z /= rest_x * c;*/
			sdata_r /= rest_x * c;
			sdata_g /= rest_x * c;
			sdata_b /= rest_x * c;
		}
		else if (blockIdx.x != block_x && blockIdx.y == block_y) {
			sdata_r /= c * rest_y;
			sdata_g /= c * rest_y;
			sdata_b /= c * rest_y;
		}
		else {
			sdata_r /= c * c;
			sdata_g /= c * c;
			sdata_b /= c * c;
		}
	}

	__syncthreads();
	// fill in the avreage values
	if (blockIdx.x != block_x || threadIdx.x < rest_x) {
		for (unsigned int j = 0; j < colPerThread; j++) {
			for (unsigned int i = 0; i < c; i++) {
				if (blockIdx.y == block_y && i >= rest_y) continue;
				if ((blockIdx.x * c + threadIdx.x + j * 1024) >= width) continue;
				unsigned int offset = (blockIdx.y * width * c) + (blockIdx.x * c + threadIdx.x + j * 1024) + (width * i);

				image[offset].x = sdata_r;
				image[offset].y = sdata_g;
				image[offset].z = sdata_b;
				/*gpu_r[offset] = sdata_r;
				gpu_g[offset] = sdata_g;
				gpu_b[offset] = sdata_b;*/
			}
		}
	}

}


/*
__global__ void avgKernel_2D(unsigned long long int* GPUred, unsigned long long int* GPUgreen, unsigned long long int* GPUblue, unsigned char* GPUavg_r, unsigned char* GPUavg_g, unsigned char* GPUavg_b, const int width, const int height, const size_t pitch, const int c) {
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	//printf("blockIdx.x=%d, blockIdx.y=%d, blockDim.x=%d, blockDim.y=%d, threadIdx.x=%d, threadIdx.y=%d, xIndex=%d, yIndex=%d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y, xIndex, yIndex);

	float output_r = 0.0f;
	float output_g = 0.0f;
	float output_b = 0.0f;

	//Make sure the current thread is inside the image bounds
	if (xIndex < width && yIndex < height) {
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			//Sum the window pixels
			for (int i = 0; i < c; i++) {
				for (int j = 0; j < c; j++) {
				//The tex2D automatically handles Out-Of-Range access.
				output_r += tex2D(texData_r, xIndex + i, yIndex + j);
				output_g += tex2D(texData_g, xIndex + i, yIndex + j);
				output_b += tex2D(texData_b, xIndex + i, yIndex + j);
			}
		}
		atomicAdd(GPUred, output_r);
		atomicAdd(GPUgreen, output_g);
		atomicAdd(GPUblue, output_b);

		output_r /= (c * c);
		output_g /= (c * c);
		output_b /= (c * c);

		//Write the averaged value to the output.
		//Transform 2D index to 1D index, because image is actually in linear memory
		//pitch = blockIdx.x * blockDim.x
		int index = yIndex * pitch + xIndex;

		//printf("blockIdx.x=%d, blockIdx.y=%d, threadIdx.x=%d, threadIdx.y=%d, xIndex=%d, yIndex=%d, pitch=%d, index=%d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, xIndex, yIndex, pitch, index);
		GPUavg_r[index] = static_cast<unsigned char>(output_r);
		GPUavg_g[index] = static_cast<unsigned char>(output_g);
		GPUavg_b[index] = static_cast<unsigned char>(output_b);
		}
	}
}
*/


/*
__global__ void fillKernel_2D(unsigned char* GPUoutput_r, unsigned char* GPUoutput_g, unsigned char* GPUoutput_b, unsigned char* GPUavg_r, unsigned char* GPUavg_g, unsigned char* GPUavg_b, const int width, const int height, const size_t pitch) {
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if (xIndex < width && yIndex < height) {
		int index = yIndex * pitch + xIndex;
		int base = (blockIdx.y * blockDim.y + 0) * pitch + (blockIdx.x * blockDim.x + 0);
		GPUoutput_r[index] = GPUavg_r[base];
		GPUoutput_g[index] = GPUavg_g[base];
		GPUoutput_b[index] = GPUavg_b[base];
	}
}
*/


int main(int argc, char *argv[]) {
	int c = 0;
	char *input_file = (char*)malloc(100);
	char *output_file = (char*)malloc(100);
	char *ppm_format = (char*)malloc(15);
	int width = 0, height = 0;
	char *header = (char*)malloc(1024);
	unsigned long long int ave_r = 0, ave_g = 0, ave_b = 0;

	if (process_command_line(argc, argv, &c, input_file, output_file, ppm_format) == FAILURE) {
		return 1;
	}
	printf("[c size] %d\n", c);
	printf("[input filename] %s\n", input_file);
	printf("[output filename] %s\n", output_file);
	printf("[formate] %s\n", ppm_format);

	// read input image file (either binary or plain text PPM)
	if (image_input(input_file, c, &width, &height, header, output_file, ppm_format) == FAILURE) {
		return 1;
	}
	printf("[image width] %d\n", width);
	printf("[image height] %d\n", height);
	printf("\n");

	// allocate cpu memory for storing the results from cuda
	unsigned char* cpu_r = (unsigned char *)malloc(sizeof(unsigned char)*(width)*(height));
	unsigned char* cpu_g = (unsigned char *)malloc(sizeof(unsigned char)*(width)*(height));
	unsigned char* cpu_b = (unsigned char *)malloc(sizeof(unsigned char)*(width)*(height));

	// execute the mosaic filter based on the mode
	switch (execution_mode) {
	case (CPU): {
		// CPU mode
		printf("======================================= CPU MODE ========================================\n");
		// calculate the average colour value
		
		cul_average_cpu(c, width, height, &ave_r, &ave_g, &ave_b);
		// Output the average colour value for the image
		printf("CPU Average image colour red = %llu, green = %llu, blue = %llu \n", ave_r, ave_g, ave_b);

		break;
	}
	case (OPENMP): {
		// OPENMP mode
		printf("====================================== OPENMP MODE ======================================\n");
		// calculate the average colour value
		cul_average_openmp(c, width, height, &ave_r, &ave_g, &ave_b);
		// Output the average colour value for the image
		printf("OPENMP Average image colour red = %llu, green = %llu, blue = %llu \n", ave_r, ave_g, ave_b);

		break;
	}
	case (CUDA): {
		// CUDA mode
		printf("======================================= CUDA MODE =======================================\n");
		transform_2D_to_1D(cpu_r, cpu_g, cpu_b, width, height);

		// calculate the average colour value
		launch_cuda_1D(c, width, height, cpu_r, cpu_g, cpu_b, &ave_r, &ave_g, &ave_b);
		//launch_cuda_2D(c, width, height, cpu_r, cpu_g, cpu_b, &ave_r, &ave_g, &ave_b);

		// Output the average colour value for the image
		printf("CUDA Average image colour red = %llu, green = %llu, blue = %llu \n", ave_r, ave_g, ave_b);
		transform_1D_to_2D(cpu_r, cpu_g, cpu_b, width, height);

		break;
	}
	case (ALL): {
		// CPU mode
		printf("======================================= CPU MODE ========================================\n");
		// CPU: calculate the average colour value
		cul_average_cpu(c, width, height, &ave_r, &ave_g, &ave_b);
		// CPU: output the average colour value for the image
		printf("CPU Average image colour red = %llu, green = %llu, blue = %llu \n\n", ave_r, ave_g, ave_b);

		// OPENMP mode
		printf("====================================== OPENMP MODE ======================================\n");
		// OPENMP: calculate the average colour value
		cul_average_openmp(c, width, height, &ave_r, &ave_g, &ave_b);
		// OPENMP: output the average colour value for the image
		printf("OPENMP Average image colour red = %llu, green = %llu, blue = %llu \n\n", ave_r, ave_g, ave_b);

		// CUDA mode
		printf("======================================= CUDA MODE =======================================\n");
		transform_2D_to_1D(cpu_r, cpu_g, cpu_b, width, height);
		// CUDA: calculate the average colour value
		launch_cuda_1D(c, width, height, cpu_r, cpu_g, cpu_b, &ave_r, &ave_g, &ave_b);
		// CUDA: output the average colour value for the image
		printf("CUDA Average image colour red = %llu, green = %llu, blue = %llu \n\n", ave_r, ave_g, ave_b);
		transform_1D_to_2D(cpu_r, cpu_g, cpu_b, width, height);

		break;
	}
	}

	//save the output image file (from last executed mode)
	if (image_output(width, height, ppm_format, header, output_file) == FAILURE) {
		return 1;
	}

	//free memory
	int k;
	for (k = 0; k < height; k++) {
		free(image_r[k]);
		free(image_g[k]);
		free(image_b[k]);
	}

	//Free CPU memory
	free(cpu_r);
	free(cpu_g);
	free(cpu_b);
	free(image_r);
	free(image_g);
	free(image_b);
	free(input_file);
	free(output_file);
	free(ppm_format);
	free(header);

	return 0;
}


void print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP, CUDA or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");
}


int process_command_line(int argc, char *argv[], int *c, char *input_file, char *output_file, char *ppm_format) {
	// limit the number of argument between 7 and 9
	if (argc != 7 && argc != 9) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}
	
	// read in the non optional command line arguments
	*c = atoi(argv[1]);
	if (*c < 1) {
		fprintf(stderr, "Error: input c cannot be less than 1...\n");
		return FAILURE;
	}
	// ensure the c is power of 2
	if (is_two_n(*c) != 1) {
		fprintf(stderr, "Error: input c is not a power of 2 number...\n");
		return FAILURE;
	}

	// read in the mode
	if (strcmp("CPU", argv[2]) == 0) execution_mode = CPU;
	else if (strcmp("OPENMP", argv[2]) == 0) execution_mode = OPENMP;
	else if (strcmp("CUDA", argv[2]) == 0) execution_mode = CUDA;
	else if (strcmp("ALL", argv[2]) == 0) execution_mode = ALL;

	// read in the input image name
	if (strcmp("-i", argv[3]) == 0) {
		if ((strstr(argv[4], ".ppm") == NULL) && (strstr(argv[4], ".PPM") == NULL)) {
			fprintf(stderr, "Error: input file shoud be a ppm image...\n");
			return FAILURE;
		}
		strcpy(input_file, argv[4]);
	}
	else {
		fprintf(stderr, "Error: Wrong program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}

	// read in the output image name
	if (strcmp("-o", argv[5]) == 0) {
		if ((strstr(argv[6], ".ppm") == NULL) && (strstr(argv[6], ".PPM") == NULL)) {
			fprintf(stderr, "Error: output file shoud be a ppm image...\n");
			return FAILURE;
		}
		strcpy(output_file, argv[6]);
	}
	else {
		fprintf(stderr, "Error: Wrong program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}

	// read in any optional part 3 arguments
	// the defult output format is PPM_BINARY
	if (argc == 9) {
		if (strcmp("-f", argv[7]) == 0) {
			strcpy(ppm_format, argv[8]);
		}
		else {
			fprintf(stderr, "Error: Wrong program arguments. Correct usage is...\n");
			print_help();
			return FAILURE;
		}
	}
	else {
		strcpy(ppm_format, "PPM_BINARY");
	}

	return SUCCESS;
}


int is_two_n(int num) {
	if ((num&(num - 1))) {
		return -1;
	}
	return 1;
}


int image_input(char *input_file, int c, int *width, int *height, char *header, char *output_file, char *ppm_format) {
	// open file
	FILE *fp = fopen(input_file, "rb");
	// the input file does not exist
	if (fp == NULL) {
		fprintf(stderr, "Error: Can't find the input file...\n");
		return FAILURE;
	}

	char *format = (char*)malloc(3);
	if (read_header(fp, c, width, height, header, output_file, ppm_format, format) == FAILURE) {
		fprintf(stderr, "Error: Can't read header...\n");
		return FAILURE;
	}

	// initialise two dimensions dynamic int array
	// allocate memory space to first dimensions
	image_r = (unsigned char **)malloc(sizeof(unsigned char *)*(*height));
	image_g = (unsigned char **)malloc(sizeof(unsigned char *)*(*height));
	image_b = (unsigned char **)malloc(sizeof(unsigned char *)*(*height));
	int k;
	// allocate memory space to second dimensions of each first dimension
	for (k = 0; k < *height; k++) {
		*(image_r + k) = (unsigned char *)malloc(sizeof(unsigned char)*(*width));
		*(image_g + k) = (unsigned char *)malloc(sizeof(unsigned char)*(*width));
		*(image_b + k) = (unsigned char *)malloc(sizeof(unsigned char)*(*width));
	}

	// allocate momory 
	unsigned char *all_input = (unsigned char *)malloc(sizeof(unsigned char)*(*width)*(*height) * 3);
	// read content infomation
	// PPM_PLAIN_TEXT format
	if (strcmp(format, "P3") == 0) {
		unsigned char buf;
		int count = 0;
		while (fscanf(fp, "%hhu", &buf) == 1) {
			all_input[count] = buf;
			count++;
		}
	}
	// PPM_BINARY format
	else if (strcmp(format, "P6") == 0) {
		// read all the binary content
		fread(all_input, sizeof(unsigned char), 3 * (*width) * (*height), fp);
	}

	int i = 0, h = -1, w = 0;;
	while (i < (*width)*(*height) * 3) {
		if (i % (*width * 3) == 0) {
			h++;
			w = 0;
		}
		// red
		if (i % 3 == 0) {
			image_r[h][w] = all_input[i];
			//printf("%d ", image_r[h][w]);
		}
		// green
		else if (i % 3 == 1) {
			image_g[h][w] = all_input[i];
			//printf("%d ", image_g[h][w]);
		}
		// blue
		else {
			image_b[h][w] = all_input[i];
			//printf("%d ", image_b[h][w]);
			w++;
		}
		i++;
	}
	// close file
	fclose(fp);
	free(all_input);
	free(format);

	return SUCCESS;
}


int read_header(FILE *fp, int c, int *width, int *height, char *header, char *output_file, char *ppm_format, char *format) {
	char input[1024] = "";

	if (strcmp(ppm_format, "PPM_PLAIN_TEXT") == 0) {
		strcpy(header, "P3\n");
	}
	else if (strcmp(ppm_format, "PPM_BINARY") == 0) {
		strcpy(header, "P6\n");
	}
	strcat(header, "# COM6521 Assignment2 - ");
	strcat(header, output_file);
	strcat(header, "\n");

	// read header infomation
	while (1) {
		// exit if reading to the end of file
		if (fgets(input, sizeof(input), fp) == NULL) {
			return FAILURE;
		}
		// exit if reading to the end line of header
		if (strncmp(input, "255", 3) == 0) {
			strcat(header, input);
			break;
		}
		// file format (either P3 or P6)
		if (strncmp(input, "P3", 2) == 0) {
			strcpy(format, "P3");
		}
		else if (strncmp(input, "P6", 2) == 0) {
			strcpy(format, "P6");
		}
		// skip if reading to command line
		else if (strncmp(input, "#", 1) == 0) {
			continue;
		}
		// first number is file width and sencond one is height
		else {
			strcat(header, input);
			char * ptr = strchr(input, ' ');
			if (ptr != NULL) {
				*height = atoi(ptr);
			}
			// width is not assigned
			if (*width == 0) {
				*width = atoi(input);
			}
			else {
				*height = atoi(input);
			}
		}
	}

	// limit c should be less than width and height 
	if (c > *width || c > *height) {
		fprintf(stderr, "Error: input c is greater than width or height...\n");
		return FAILURE;
	}
	return SUCCESS;
}


void cul_average_cpu(int c, int width, int height, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b) {
	clock_t begin, end;
	float mseconds;

	// starting timing here
	begin = clock();

	// initialise the results
	unsigned long long int red = 0, green = 0, blue = 0;
	*ave_r = 0, *ave_g = 0, *ave_b = 0;
	int i = 0, j = 0, k = 0, l = 0;
	for (i = 0; i < height; i += c) {
		for (j = 0; j < width; j += c) {
			unsigned long long int sum_r = 0, sum_g = 0, sum_b = 0;
			unsigned long long int count = 0;
			// sum the values in a cell
			for (k = i; k < (i + c) && k < height; k++) {
				for (l = j; l < (j + c) && l < width; l++) {
					count++;
					sum_r += image_r[k][l];
					sum_g += image_g[k][l];
					sum_b += image_b[k][l];
				}
			}
			// replace the origin values by the cell average
			for (k = i; k < (i + c) && k < height; k++) {
				for (l = j; l < (j + c) && l < width; l++) {
					image_r[k][l] = (unsigned char)(sum_r / count);
					image_g[k][l] = (unsigned char)(sum_g / count);
					image_b[k][l] = (unsigned char)(sum_b / count);
				}
			}
			red += sum_r;
			green += sum_g;
			blue += sum_b;
			
		}
	}
	*ave_r = red / (height * width);
	*ave_g = green / (height * width);
	*ave_b = blue / (height * width);

	// end timing here
	end = clock();
	mseconds = (end - begin) * 1000 / (float)CLOCKS_PER_SEC;
	printf("CPU mode execution time took %d s and %d ms\n", (int)mseconds / 1000, (int)mseconds % 1000);
}


void cul_average_openmp(int c, int width, int height, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b) {
	clock_t begin, end;
	float mseconds;

	// starting timing here
	begin = clock();

	// initialise the results
	unsigned long long int red = 0, green = 0, blue = 0;
	int i;
#pragma omp parallel for
	for (i = 0; i < height; i += c) {
		int j = 0;
		//#pragma omp parallel for
#pragma omp parallel for reduction(+: red, green, blue)
		for (j = 0; j < width; j += c) {
			int k, l;
			unsigned long long int sum_r = 0, sum_g = 0, sum_b = 0, count = 0;
			// sum the values in a cell
			for (k = i; k < (i + c) && k < height; k++) {
				for (l = j; l < (j + c) && l < width; l++) {
					count++;
					sum_r += image_r[k][l];
					sum_g += image_g[k][l];
					sum_b += image_b[k][l];
				}
			}
			// replace the origin values by the cell average
			for (k = i; k < (i + c) && k < height; k++) {
				for (l = j; l < (j + c) && l < width; l++) {
					image_r[k][l] = (unsigned char)(sum_r / count);
					image_g[k][l] = (unsigned char)(sum_g / count);
					image_b[k][l] = (unsigned char)(sum_b / count);
				}
			}
			//#pragma omp critical
			//{
			red += sum_r;
			green += sum_g;
			blue += sum_b;
			//}
		}
	}
	*ave_r = red / (height * width);
	*ave_g = green / (height * width);
	*ave_b = blue / (height * width);

	// end timing here
	end = clock();
	mseconds = (end - begin) * 1000 / (float)CLOCKS_PER_SEC;
	printf("OPENMP mode execution time took %d s and %d ms\n", (int)mseconds / 1000, (int)mseconds % 1000);
}


void launch_cuda_1D(int c, int width, int height, unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b) {
	hipEvent_t start, stop;
	hipEvent_t k_start, k_stop;
	float ms, mseconds;

	uchar3 *d_image;
	uchar3 *h_image;
	h_image = (uchar3*)malloc(sizeof(uchar3)*(width)*(height));
	for (int i = 0; i < width*height; i++) {
		h_image[i].x = cpu_r[i];
		h_image[i].y = cpu_g[i];
		h_image[i].z = cpu_b[i];
	}

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&k_start);
	hipEventCreate(&k_stop);

	// starting timing here
	hipEventRecord(start, 0);

	// initalise the values of rgb average
	*ave_r = 0, *ave_g = 0, *ave_b = 0;

	//Declare GPU pointer
	//unsigned char *gpu_r, *gpu_g, *gpu_b;

	// allocate memory on the GPU
	hipMalloc((void**)&d_image, sizeof(uchar3)*(width)*(height));
	/*hipMalloc((void**)&gpu_r, sizeof(unsigned char)*(width)*(height));
	hipMalloc((void**)&gpu_g, sizeof(unsigned char)*(width)*(height));
	hipMalloc((void**)&gpu_b, sizeof(unsigned char)*(width)*(height));*/
	checkCUDAError("CUDA malloc");

	// For device variables
	hipMemcpyToSymbol(HIP_SYMBOL(average_r), ave_r, sizeof(unsigned long long int));
	hipMemcpyToSymbol(HIP_SYMBOL(average_g), ave_g, sizeof(unsigned long long int));
	hipMemcpyToSymbol(HIP_SYMBOL(average_b), ave_b, sizeof(unsigned long long int));

	// transfer memory from the host to device
	hipMemcpy(d_image, h_image, sizeof(uchar3)*(width)*(height), hipMemcpyHostToDevice);
	/*hipMemcpy(gpu_r, cpu_r, sizeof(unsigned char)*(width)*(height), hipMemcpyHostToDevice);
	hipMemcpy(gpu_g, cpu_g, sizeof(unsigned char)*(width)*(height), hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, cpu_b, sizeof(unsigned char)*(width)*(height), hipMemcpyHostToDevice);*/
	checkCUDAError("CUDA memcpy to device");

	int block_x = (width % c == 0) ? width / c : width / c + 1;
	int block_y = (height % c == 0) ? height / c : height / c + 1;
	int thread = (c > 1024) ? 1024 : c;
	//cuda layout and execution
	dim3 threadsPerBlock(thread, 1, 1);
	dim3 blocksPerGrid(block_x, block_y, 1);

	hipEventRecord(k_start, 0);
	// lauch kernel
	//avgKernel_1D << <blocksPerGrid, threadsPerBlock >> > (gpu_r, gpu_g, gpu_b, width, height, c);
	avgKernel_1D << <blocksPerGrid, threadsPerBlock >> > (d_image, width, height, c);
	
	hipEventRecord(k_stop, 0);
	hipEventSynchronize(k_stop);
	hipEventElapsedTime(&ms, k_start, k_stop);
	printf("CUDA mode execution time only for kernel took %f ms\n", ms);
	

	hipMemcpyFromSymbol(ave_r, HIP_SYMBOL(average_r), sizeof(unsigned long long int));
	hipMemcpyFromSymbol(ave_g, HIP_SYMBOL(average_g), sizeof(unsigned long long int));
	hipMemcpyFromSymbol(ave_b, HIP_SYMBOL(average_b), sizeof(unsigned long long int));

	// transfer memory from the device to device
	hipMemcpy(h_image, d_image, sizeof(uchar3)*(width)*(height), hipMemcpyDeviceToHost);
	/*hipMemcpy(cpu_r, gpu_r, sizeof(unsigned char)*(width)*(height), hipMemcpyDeviceToHost);
	hipMemcpy(cpu_g, gpu_g, sizeof(unsigned char)*(width)*(height), hipMemcpyDeviceToHost);
	hipMemcpy(cpu_b, gpu_b, sizeof(unsigned char)*(width)*(height), hipMemcpyDeviceToHost);*/
	checkCUDAError("CUDA memcpy from device");

	*ave_r /= width * height;
	*ave_g /= width * height;
	*ave_b /= width * height;

	// end timing here
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mseconds, start, stop);
	checkCUDAError("timmer");
	printf("CUDA mode execution time took %d s and %f ms\n", (int)mseconds / 1000, mseconds);

	for (int i = 0; i < width*height; i++) {
		cpu_r[i] = h_image[i].x;
		cpu_g[i] = h_image[i].y;
		cpu_b[i] = h_image[i].z;
	}

	// release GPU memory
	hipFree(d_image);
	/*hipFree(gpu_r);
	hipFree(gpu_g);
	hipFree(gpu_b);*/

	free(h_image);

	// cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
}


/*
void launch_cuda_2D(int c, int width, int height, unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, unsigned long long int *ave_r, unsigned long long int *ave_g, unsigned long long int *ave_b) {
	hipEvent_t start, stop;
	float mseconds;

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// starting timing here
	hipEventRecord(start, 0);

	//Declare GPU pointer
	unsigned char *GPUinput_r, *GPUinput_g, *GPUinput_b, *GPUavg_r, *GPUavg_g, *GPUavg_b, *GPUoutput_r, *GPUoutput_g, *GPUoutput_b;
	unsigned long long int CPUred = 0, CPUgreen = 0, CPUblue = 0, *GPUred, *GPUgreen, *GPUblue;


	//Allocate 2D memory on GPU. Also known as Pitch Linear Memory
	size_t gpu_image_pitch = 0;
	hipMalloc((void**)&GPUred, sizeof(int));
	hipMalloc((void**)&GPUgreen, sizeof(int));
	hipMalloc((void**)&GPUblue, sizeof(int));
	hipMallocPitch<unsigned char>(&GPUinput_r, &gpu_image_pitch, width * sizeof(unsigned char), height);
	hipMallocPitch<unsigned char>(&GPUinput_g, &gpu_image_pitch, width * sizeof(unsigned char), height);
	hipMallocPitch<unsigned char>(&GPUinput_b, &gpu_image_pitch, width * sizeof(unsigned char), height);

	hipMallocPitch<unsigned char>(&GPUavg_r, &gpu_image_pitch, width * sizeof(unsigned long long int), height);
	hipMallocPitch<unsigned char>(&GPUavg_g, &gpu_image_pitch, width * sizeof(unsigned long long int), height);
	hipMallocPitch<unsigned char>(&GPUavg_b, &gpu_image_pitch, width * sizeof(unsigned long long int), height);

	hipMallocPitch<unsigned char>(&GPUoutput_r, &gpu_image_pitch, width * sizeof(unsigned char), height);
	hipMallocPitch<unsigned char>(&GPUoutput_g, &gpu_image_pitch, width * sizeof(unsigned char), height);
	hipMallocPitch<unsigned char>(&GPUoutput_b, &gpu_image_pitch, width * sizeof(unsigned char), height);

	//Copy data from host to device.
	hipMemcpy(GPUred, &CPUred, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	hipMemcpy(GPUgreen, &CPUgreen, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	hipMemcpy(GPUblue, &CPUblue, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	hipMemcpy2D(GPUinput_r, gpu_image_pitch, cpu_r, width * sizeof(unsigned char), width * sizeof(unsigned char), height, hipMemcpyHostToDevice);
	hipMemcpy2D(GPUinput_g, gpu_image_pitch, cpu_g, width * sizeof(unsigned char), width * sizeof(unsigned char), height, hipMemcpyHostToDevice);
	hipMemcpy2D(GPUinput_b, gpu_image_pitch, cpu_b, width * sizeof(unsigned char), width * sizeof(unsigned char), height, hipMemcpyHostToDevice);

	//Bind the image to the texture. Now the kernel will read the input image through the texture cache.
	//Use tex2D function to read the image
	hipBindTexture2D(NULL, texData_r, GPUinput_r, width * sizeof(unsigned char), height, gpu_image_pitch);
	hipBindTexture2D(NULL, texData_g, GPUinput_g, width * sizeof(unsigned char), height, gpu_image_pitch);
	hipBindTexture2D(NULL, texData_b, GPUinput_b, width * sizeof(unsigned char), height, gpu_image_pitch);

	// Set the behavior of tex2D for out-of-range image reads.
	texData_r.addressMode[0] = texData_r.addressMode[1] = hipAddressModeBorder;
	texData_g.addressMode[0] = texData_g.addressMode[1] = hipAddressModeBorder;
	texData_b.addressMode[0] = texData_b.addressMode[1] = hipAddressModeBorder;

	dim3 threadsPerBlock(c, c, 1);
	dim3 blocksPerGrid;
	blocksPerGrid.x = (width + threadsPerBlock.x - 1) / threadsPerBlock.x;  //< Greater than or equal to image width
	blocksPerGrid.y = (height + threadsPerBlock.y - 1) / threadsPerBlock.y; //< Greater than or equal to image height

	//Launch the kernel
	avgKernel_2D << <blocksPerGrid, threadsPerBlock >> >(GPUred, GPUgreen, GPUblue, GPUavg_r, GPUavg_g, GPUavg_b, width, height, gpu_image_pitch, c);
	fillKernel_2D << <blocksPerGrid, threadsPerBlock >> >(GPUoutput_r, GPUoutput_g, GPUoutput_b, GPUavg_r, GPUavg_g, GPUavg_b, width, height, gpu_image_pitch);

	//Copy the results back to CPU
	hipMemcpy(&CPUred, GPUred, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(&CPUgreen, GPUgreen, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(&CPUblue, GPUblue, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy2D(cpu_r, width, GPUoutput_r, gpu_image_pitch, width * sizeof(unsigned char), height, hipMemcpyDeviceToHost);
	hipMemcpy2D(cpu_g, width, GPUoutput_g, gpu_image_pitch, width * sizeof(unsigned char), height, hipMemcpyDeviceToHost);
	hipMemcpy2D(cpu_b, width, GPUoutput_b, gpu_image_pitch, width * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

	*ave_r = CPUred / (width* height);
	*ave_g = CPUgreen / (width* height);
	*ave_b = CPUblue / (width* height);

	//Release the texture
	hipUnbindTexture(texData_r);
	hipUnbindTexture(texData_g);
	hipUnbindTexture(texData_b);

	//Free GPU memory
	hipFree(GPUinput_r);
	hipFree(GPUinput_g);
	hipFree(GPUinput_b);
	hipFree(GPUoutput_r);
	hipFree(GPUoutput_g);
	hipFree(GPUoutput_b);
	hipFree(GPUavg_r);
	hipFree(GPUavg_g);
	hipFree(GPUavg_b);
	hipFree(GPUred);
	hipFree(GPUgreen);
	hipFree(GPUblue);

	// end timing here
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mseconds, start, stop);
	checkCUDAError("timmer");
	printf("CUDA mode execution time took 0 s and %f ms\n", mseconds);
}*/



void transform_2D_to_1D(unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, int width, int height) {
	int count = 0;
	// transfer 2D array to 1D
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			cpu_r[count] = image_r[i][j];
			cpu_g[count] = image_g[i][j];
			cpu_b[count] = image_b[i][j];
			count++;
		}
	}
}


void transform_1D_to_2D(unsigned char* cpu_r, unsigned char* cpu_g, unsigned char* cpu_b, int width, int height) {
	long long count = 0;
	// transfer 1D array to 2D
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			image_r[i][j] = cpu_r[count];
			image_g[i][j] = cpu_g[count];
			image_b[i][j] = cpu_b[count];
			count++;
		}
	}
}


int image_output(int width, int height, char *ppm_format, char *header, char *output_file) {
	if (strcmp(ppm_format, "PPM_BINARY") == 0) {
		// open file
		FILE *fp = fopen(output_file, "wb");
		// the output file does not exist
		if (fp == NULL) {
			fprintf(stderr, "Error: Can't find the output file...\n");
			return FAILURE;
		}
		// write header information
		fprintf(fp, "%s", header);
		unsigned char *all_output = (unsigned char *)malloc(sizeof(unsigned char)*width*height * 3);
		int i, j, k = 0;
		for (i = 0; i < height; i++) {
			for (j = 0; j < width; j++) {
				// red
				all_output[k] = image_r[i][j];
				k++;
				// green
				all_output[k] = image_g[i][j];
				k++;
				// blue
				all_output[k] = image_b[i][j];
				k++;
			}
		}
		// write all information
		fwrite(all_output, sizeof(unsigned char), 3 * width*height, fp);
		// close file
		fclose(fp);
		free(all_output);
	}
	else if (strcmp(ppm_format, "PPM_PLAIN_TEXT") == 0) {
		// open file
		FILE *fp = fopen(output_file, "w");
		// the output file does not exist
		if (fp == NULL) {
			fprintf(stderr, "Error: Can't find the output file...\n");
			return FAILURE;
		}
		// write header information
		fputs(header, fp);
		int i, j;
		char out_string[4];
		for (i = 0; i < height; i++) {
			for (j = 0; j < width; j++) {
				// red
				sprintf(out_string, "%u", image_r[i][j]);
				fputs(out_string, fp);
				fputc(' ', fp);
				// green
				sprintf(out_string, "%u", image_g[i][j]);
				fputs(out_string, fp);
				fputc(' ', fp);
				// blue
				sprintf(out_string, "%u", image_b[i][j]);
				fputs(out_string, fp);

				if (j == (width - 1)) continue;
				fputc('\t', fp);
			}
			// move to new line
			if (i == (height - 1)) continue;
			fputc('\n', fp);
		}
		// close file
		fclose(fp);
	}

	return SUCCESS;
}


void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}